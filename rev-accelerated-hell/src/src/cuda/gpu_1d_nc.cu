
#include <hip/hip_runtime.h>
extern "C" __global__ void a(const int64_t* vals, const int64_t* table, int64_t* result, uint n, int64_t modulus) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    // Dot product
    int64_t dp = 0;
    for (uint j = 0; j < n; j++) dp += table[i * j % n] * vals[j] % modulus;
    dp %= modulus;
    result[i] = dp;
}

extern "C" __global__ void b(int64_t* p1, const int64_t* p2, uint degree, int64_t modulus) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < degree) {
        p1[i] += p2[i];
        p1[i] %= modulus;
    }
}

extern "C" __global__ void c(int64_t* p1, const int64_t* p2, uint degree, int64_t modulus) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < degree) {
        p1[i] *= p2[i];
        p1[i] %= modulus;
    }
}

extern "C" __global__ void d(int64_t* poly, int64_t* phi, uint n, int64_t inverse_n, int64_t modulus) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        poly[i] *= inverse_n * phi[i] % modulus;
        poly[i] %= modulus;
    }
}